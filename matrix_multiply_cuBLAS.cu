﻿#include <iostream>
#include <vector>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>

void matmul_cublas(const float *A, const float *B, float *C, int M, int N, int K)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // cuBLAS 默认是列主序（Fortran-style）
    // 所以我们计算 C = A * B 相当于调用 GEMM(B, A)
    // 即: C(M×K) = A(M×N) * B(N×K)
    // 需要调整顺序：C = Bᵗ * Aᵗ

    hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, M, N,              // C = (K×M) matrix
        &alpha,
        B, K,                 // B: K×N
        A, N,                 // A: N×M
        &beta,
        C, K                  // C: K×M
    );

    hipblasDestroy(handle);
}

int main()
{
    int M = 2048, N = 1024, K = 2048;

    std::vector<float> A(M * N), B(N * K), C(M * K, 0.0f);

    // 初始化矩阵
    for (int i = 0; i < M * N; ++i) A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < N * K; ++i) B[i] = static_cast<float>(rand()) / RAND_MAX;

    // 分配显存
    float *dA, *dB, *dC;
    hipMalloc(&dA, M * N * sizeof(float));
    hipMalloc(&dB, N * K * sizeof(float));
    hipMalloc(&dC, M * K * sizeof(float));

    hipMemcpy(dA, A.data(), M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B.data(), N * K * sizeof(float), hipMemcpyHostToDevice);

    // 计时
    auto start = std::chrono::high_resolution_clock::now();

    matmul_cublas(dA, dB, dC, M, N, K);

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    double elapsed = std::chrono::duration<double, std::milli>(end - start).count();
    std::cout << "cuBLAS time: " << elapsed << " ms" << std::endl;

    hipMemcpy(C.data(), dC, M * K * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}
